#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <hip/hip_math_constants.h>

using namespace optix;

rtDeclareVariable(int, fill_surface_interaction, , );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(void *, accel, , );
rtDeclareVariable(unsigned long long, shape_ptr, , );
rtDeclareVariable(unsigned int, launch_index, rtLaunchIndex, );

rtDeclareVariable(float3, p, attribute p, );
rtDeclareVariable(float2, uv, attribute uv, );
rtDeclareVariable(float3, ns, attribute ns, );
rtDeclareVariable(float3, ng, attribute ng, );
rtDeclareVariable(float3, dp_du, attribute dp_du, );
rtDeclareVariable(float3, dp_dv, attribute dp_dv, );
rtDeclareVariable(Ray, ray, rtCurrentRay,);

rtBuffer<bool> in_mask;

rtBuffer<float> in_ox, in_oy, in_oz,
                in_dx, in_dy, in_dz,
                in_mint, in_maxt;

rtBuffer<float> out_t, out_u, out_v, out_ng_x, out_ng_y,
                out_ng_z, out_ns_x, out_ns_y, out_ns_z,
                out_p_x, out_p_y, out_p_z,
                out_dp_du_x, out_dp_du_y, out_dp_du_z,
                out_dp_dv_x, out_dp_dv_y, out_dp_dv_z;

rtBuffer<unsigned long long> out_shape_ptr;

rtBuffer<uint32_t> out_primitive_id;

rtBuffer<bool> out_hit;

struct PerRayData { };

RT_PROGRAM void ray_gen_closest() {
    float3 ro = make_float3(in_ox[launch_index],
                            in_oy[launch_index],
                            in_oz[launch_index]),
           rd = make_float3(in_dx[launch_index],
                            in_dy[launch_index],
                            in_dz[launch_index]);
    float  mint = in_mint[launch_index],
           maxt = in_maxt[launch_index];

    if (!in_mask[launch_index]) {
        out_shape_ptr[launch_index] = 0;
        out_t[launch_index] = HIP_INF_F;
    } else {
        PerRayData prd;
        Ray ray = make_Ray(ro, rd, 0, mint, maxt);
        rtTrace(top_object, ray, prd);
    }
}

RT_PROGRAM void ray_gen_any() {
    float3 ro = make_float3(in_ox[launch_index],
                            in_oy[launch_index],
                            in_oz[launch_index]),
           rd = make_float3(in_dx[launch_index],
                            in_dy[launch_index],
                            in_dz[launch_index]);
    float  mint = in_mint[launch_index],
           maxt = in_maxt[launch_index];

    Ray ray = make_Ray(ro, rd, 0, mint, maxt);

    if (!in_mask[launch_index]) {
        out_hit[launch_index] = false;
    } else {
        PerRayData prd;
        rtTrace(top_object, ray, prd, RT_VISIBILITY_ALL,
                RT_RAY_FLAG_TERMINATE_ON_FIRST_HIT);
    }
}

__device__ inline float squared_norm(float3 v) {
    return dot(v, v);
}

RT_PROGRAM void ray_hit() {
    if (out_hit.size() > 0) {
        out_hit[launch_index] = true;
    } else {
        out_shape_ptr[launch_index] = shape_ptr;

        out_primitive_id[launch_index] = rtGetPrimitiveIndex();

        out_u[launch_index] = uv.x;
        out_v[launch_index] = uv.y;

        out_p_x[launch_index] = p.x;
        out_p_y[launch_index] = p.y;
        out_p_z[launch_index] = p.z;

        if (fill_surface_interaction == 1) {
            out_ng_x[launch_index] = ng.x;
            out_ng_y[launch_index] = ng.y;
            out_ng_z[launch_index] = ng.z;

            out_ns_x[launch_index] = ns.x;
            out_ns_y[launch_index] = ns.y;
            out_ns_z[launch_index] = ns.z;

            out_dp_du_x[launch_index] = dp_du.x;
            out_dp_du_y[launch_index] = dp_du.y;
            out_dp_du_z[launch_index] = dp_du.z;

            out_dp_dv_x[launch_index] = dp_dv.x;
            out_dp_dv_y[launch_index] = dp_dv.y;
            out_dp_dv_z[launch_index] = dp_dv.z;
        }

        out_t[launch_index] = sqrt(squared_norm(p - ray.origin) / squared_norm(ray.direction));
    }
}

RT_PROGRAM void ray_miss() {
    if (out_hit.size() > 0) {
        out_hit[launch_index] = false;
    } else {
        out_shape_ptr[launch_index] = 0;
        out_t[launch_index] = HIP_INF_F;
    }
}

RT_PROGRAM void ray_err() {
    rtPrintExceptionDetails();
}
