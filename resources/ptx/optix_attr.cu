#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(int, fill_surface_interaction, , );

rtDeclareVariable(float3, p, attribute p, );
rtDeclareVariable(float2, uv, attribute uv, );
rtDeclareVariable(float3, ns, attribute ns, );
rtDeclareVariable(float3, ng, attribute ng, );
rtDeclareVariable(float3, dp_du, attribute dp_du, );
rtDeclareVariable(float3, dp_dv, attribute dp_dv, );

rtBuffer<uint3> faces;
rtBuffer<float3> vertex_positions;
rtBuffer<float3> vertex_normals;
rtBuffer<float2> vertex_texcoords;

__device__ void coordinate_system(float3 n, float3 &x, float3 &y) {
    /* Based on "Building an Orthonormal Basis, Revisited" by
       Tom Duff, James Burgess, Per Christensen,
       Christophe Hery, Andrew Kensler, Max Liani,
       and Ryusuke Villemin (JCGT Vol 6, No 1, 2017) */

    float s = copysignf(1.f, n.z),
          a = -1.f / (s + n.z),
          b = n.x * n.y * a;

    x = make_float3(n.x * n.x * a * s + 1.f, b * s, -n.x * s);
    y = make_float3(b, s + n.y * n.y * a, -n.y);
}

RT_PROGRAM void ray_attr() {
    uv = rtGetTriangleBarycentrics();

    float b0 = 1.f - uv.x - uv.y,
          b1 = uv.x,
          b2 = uv.y;

    uint3 face = faces[rtGetPrimitiveIndex()];

    float3 p0 = vertex_positions[face.x],
           p1 = vertex_positions[face.y],
           p2 = vertex_positions[face.z];

    float3 dp0 = p1 - p0,
           dp1 = p2 - p0;

    p = p0 * b0 + p1 * b1 + p2 * b2;

    if (fill_surface_interaction == 1) {
        ng = normalize(cross(dp0, dp1));
        coordinate_system(ng, dp_du, dp_dv);

        if (vertex_normals.size() > 0) {
            float3 n0 = vertex_normals[face.x],
                   n1 = vertex_normals[face.y],
                   n2 = vertex_normals[face.z];

            ns = n0 * b0 + n1 * b1 + n2 * b2;
        } else {
            ns = ng;
        }

        if (vertex_texcoords.size() > 0) {
            float2 t0 = vertex_texcoords[face.x],
                   t1 = vertex_texcoords[face.y],
                   t2 = vertex_texcoords[face.z];

            uv = t0 * b0 + t1 * b1 + t2 * b2;

            float2 dt0 = t1 - t0, dt1 = t2 - t0;
            float det = dt0.x * dt1.y - dt0.y * dt1.x;

            if (det != 0.f) {
                float inv_det = 1.f / det;
                dp_du = ( dt1.y * dp0 - dt0.y * dp1) * inv_det;
                dp_dv = (-dt1.x * dp0 + dt0.x * dp1) * inv_det;
            }
        }
    }
}
